#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h> 
#define SIZE 1<<10
#define SHMEM_SIZE 1024 * 8
#define PI 3.14159265

__global__
void mult(int n, double *x, double *y, double *z)
{
  //extern __shared__ int sdata[]; 
  //int i = blockIdx.x*blockDim.x + threadIdx.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride){
    z[i] = x[i] * y[i];
  }
}

__global__ void sum_reduction(double *v, double *v_r) {
	// Allocate shared memory
	extern __shared__ double partial_sum[];

	// Calculate thread ID
	//int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements AND do first add of reduction
	// Vector now 2x as long as number of threads, so scale i
	int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	// Store first partial result instead of just the elements
	partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];
	__syncthreads();

	// Start at 1/2 block stride and divide by two each iteration
	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
		// Each thread does work unless it is further than the stride
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = partial_sum[0];
	}
}


double CPU_angle(double * x, double * y, int n);
double CPU_angle(double * x, double * y, int n){
  double sum1 = 0;
  double sum2 = 0;
  double sum3 = 0;

  for(int i = 0; i < n; i++){
      sum1 += ( x[i] * y[i] );
      sum2 += ( x[i] * x[i] );
      sum3 += ( y[i] * y[i] );
  }

  double val = 180.0 / PI;
	double det = sqrt(sum2)*sqrt(sum3) ;
  double result = atan2( det, sum1 ) *val;
  return result;
}

int main(int argc, char *argv[]) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int n = 1;
  int b_size;

  if(argv[1] != NULL){
    n = atoi(argv[1]);
    
  }
  if(argv[2] != NULL){
    #undef SIZE 
    #define SIZE  atoi(argv[2])
    #undef SHMEM_SIZE
    #define SHMEM_SIZE SIZE*4

  }
  FILE *myFile;
  //printf("%s",argv[3]);
  if(argv[3] !=NULL){
    
    
    myFile = fopen(argv[3], "r");
    fscanf(myFile,"%d",&n);
    n = n/2;
    //nvccprintf("aaaaaaaaaaaaaaaaaaaaaaaaaaaaaa\n");
  }
  b_size = (n+SIZE-1)/SIZE;
  
  printf("Info\n");
  printf("——————\n");
  printf("Number of elements: %d\n",n);
  printf("Number of threads per block: %d\n",SIZE);
  printf("Number of blocks will be created: %d\n\n",b_size);
  
  size_t bytes = n * sizeof(double);
  
  //printf("%d",b_size);
  
  hipEventRecord(start);
  double *x, *y,*d_x, *d_y;
  
  x = (double*)malloc(bytes);
  y = (double*)malloc(bytes);
  
  double* multiplications, * square1, * square2,* d_multiplications, * d_square1, * d_square2;
  
  multiplications = (double*)malloc(bytes);
  square1 = (double*)malloc(bytes);
  square2 = (double*)malloc(bytes);

  double *uv,*d_uv;
  double *IuI, *IvI, *d_IuI, *d_IvI; 
  uv = (double*)malloc(bytes);
  IuI = (double*)malloc(bytes);
  IvI = (double*)malloc(bytes);
  
  hipMalloc(&d_x, bytes); 
  hipMalloc(&d_y, bytes);
  hipMalloc(&d_multiplications, bytes); 
  hipMalloc(&d_square1, bytes);
  hipMalloc(&d_square2, bytes); 
  hipMalloc(&d_IuI,bytes);
  hipMalloc(&d_IvI,bytes);
  hipMalloc(&d_uv,bytes);
  for (int i = 0; i < n; i++) {
    multiplications[i] = 0;
    square1[i]=0;
    square2[i]=0;
    IuI[i]= 0;
    IvI[i]= 0;
    uv[i] = 0;
  }
  if(argv[3] != NULL){
    for (size_t i = 0; i < n; i++){
      fscanf(myFile,"%lf",&x[i]);
    }
    for (size_t i = 0; i < n; i++){
      fscanf(myFile,"%lf",&y[i]);
    }
    fclose(myFile); 
     
  }else{
    
    for (int i = 0; i < n; i++) {
      //srand(time(0));
      x[i] = rand() % 100;
      //printf("%d ", x[i]);
      y[i] = rand() % 100;
      
    }  
  }

  
  
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  

  printf("Time\n");
  printf("——————\n");
  printf("Time for the array generation :%f ms\n",milliseconds);

  hipEventRecord(start);
  double cpu_result = CPU_angle(x,y,n);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time for the CPU function : %f ms\n",milliseconds);

  hipEventRecord(start);

  hipMemcpy(d_x, x, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_multiplications, multiplications, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_square1, square1, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_square2, square2, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_IuI,IuI,1*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_IvI,IvI,1*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_uv,uv,1*sizeof(double), hipMemcpyHostToDevice);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time for the Host toDevice transfer : %f ms\n",milliseconds);

  hipEventRecord(start);

  mult<<<b_size, SIZE>>>(n, d_x, d_y,d_multiplications);
  mult<<<b_size, SIZE>>>(n, d_x, d_x, d_square1);
  mult<<<b_size, SIZE>>>(n, d_y, d_y, d_square2);


	// Original vector and result vector
	


	// TB Size
	int TB_SIZE = SIZE;

	// Grid Size (cut in half) (No padding)
	int GRID_SIZE = (n / (TB_SIZE) / 2) ;

	// Call kernel
	sum_reduction << <GRID_SIZE, TB_SIZE, TB_SIZE*8 >> > (d_multiplications, d_uv);

  sum_reduction << <1, TB_SIZE, TB_SIZE*8 >> > (d_uv, d_uv);
    

  sum_reduction << <GRID_SIZE, TB_SIZE, TB_SIZE*8 >> > (d_square1, d_IuI);

  sum_reduction << <1, TB_SIZE, TB_SIZE*8 >> > (d_IuI, d_IuI);
    
    
  sum_reduction << <GRID_SIZE, TB_SIZE, TB_SIZE*8 >> > (d_square2, d_IvI);

	sum_reduction << <1, TB_SIZE, TB_SIZE*8 >> > (d_IvI, d_IvI);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds2 = 0;
  hipEventElapsedTime(&milliseconds2, start, stop);
  printf("Time for the kernel execution :%f ms\n",milliseconds2);
  
  // Copy to host;
  hipEventRecord(start);
	hipMemcpy(uv,d_uv, bytes, hipMemcpyDeviceToHost);
  hipMemcpy(IuI,d_IuI, bytes, hipMemcpyDeviceToHost);
  hipMemcpy(IvI,d_IvI, bytes, hipMemcpyDeviceToHost);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds3 = 0;
  hipEventElapsedTime(&milliseconds3, start, stop);
  printf("Time for the Deviceto Host transfer :%f ms\n",milliseconds3);
  printf("Total execution time for GPU: %f ms\n\n",milliseconds+milliseconds2+milliseconds3);
  double val = 180.0 / PI;
	double det = sqrt(IuI[0])*sqrt(IvI[0]) ;
  double gpu_result = atan2(det, uv[0] ) *val;

  
  
  printf("Results\n");
  printf("——————\n");
  printf("CPU Result: %lf\n", cpu_result);
  printf("GPU Result: %lf\n\n", gpu_result);
  
  hipFree(d_IuI);
  hipFree(d_IvI);
  hipFree(d_multiplications);
  hipFree(d_square1);
  hipFree(d_square2);
  hipFree(d_uv);
  hipFree(d_x);
  hipFree(d_y);

  free(x);
  free(y);
  free(IuI);
  free(IvI);
  free(uv);
  free(multiplications);
  free(square1);
  free(square2);


	return 0;
}
